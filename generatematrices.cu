#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <math.h>
#include <fstream>

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CUBLA_CALL(x) do { if((x)!=HIPBLAS_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

void matTofile(int m, int n, hiprandGenerator_t rng);

struct Abs: public thrust::unary_function<double, double>
{
    __host__ __device__ double operator()(double x)
    {
        return abs(x);
    }
};

struct Inv: public thrust::unary_function<double, double>
{
    __host__ __device__ double operator()(double x)
    {
        return (double) 1.0 / x;
    }
};

template<typename T>
struct MulC: public thrust::unary_function<T, T>
{
    T C;
    __host__ __device__ MulC(T c) :
        C(c)
    {
    }
    __host__ __device__ T operator()(T x)
    {
        return x * C;
    }
};

template<typename T>
struct line2col: public thrust::unary_function<T, T>
{
    T C;
    __host__ __device__ line2col(T C) :
            C(C)
    {
    }

    __host__ __device__ T operator()(T i)
    {
        return i / C;
    }
};

int main()
{
    
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    hipblasHandle_t hd;
    hiprandGenerator_t rng;
    hipblasCreate(&hd);
    hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT);
    const size_t m =19600, n = 19600;

    int times = 214200 / 80;

    float size = m*n*5 * sizeof( float );
    std::cout << "size: " << size << ", times: " << times <<std::endl;
    std::cout << time(NULL) << std::endl;
    for(int i = 0; i < times; ++i){
        matTofile(m,n,rng);
    }
    std::cout << time(NULL) << std::endl;
    hiprandDestroyGenerator(rng);
    hipblasDestroy(hd);

    return 0;
}

void matTofile(int m, int n, hiprandGenerator_t rng){
    std::ofstream ofile("rands_out.txt");
    thrust::device_vector<double> A(m * n);
    thrust::device_vector<double> C(m * n);
    thrust::device_vector<double> sum1(1 * n);
    thrust::device_vector<double> sum2(1 * n);
    thrust::device_vector<double> one(m * n, 1);

    double* pA = thrust::raw_pointer_cast(&A[0]);
    double* pSum1 = thrust::raw_pointer_cast(&sum1[0]);
    double* pSum2 = thrust::raw_pointer_cast(&sum2[0]);
    double* pOne = thrust::raw_pointer_cast(&one[0]);

    hiprandGenerateUniformDouble(rng, pA, A.size());

    const int count = 2;
    
    for (int i = 0; i < count; i++)
    {
        thrust::reduce_by_key(
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), line2col<int>(m)),
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), line2col<int>(m)) + A.size(),
                thrust::make_transform_iterator(A.begin(), Abs()),
                thrust::make_discard_iterator(),
                sum2.begin());
        thrust::transform(
                A.begin(), A.end(),
                thrust::make_permutation_iterator(
                        sum2.begin(),
                        thrust::make_transform_iterator(thrust::make_counting_iterator(0), line2col<int>(m))),
                C.begin(),
                thrust::divides<double>());
    }

    // thrust::copy(C.begin(), C.end(), std::ostream_iterator<double>(ofile, ","));

}