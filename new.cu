#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <math.h>

struct Exp: public thrust::unary_function<double, double>
{
    __host__ __device__ double operator()(double x)
    {
        return exp(x);
    }
};

struct Inv: public thrust::unary_function<double, double>
{
    __host__ __device__ double operator()(double x)
    {
        return (double) 1.0 / x;
    }
};

template<typename T>
struct MulC: public thrust::unary_function<T, T>
{
    T C;
    __host__ __device__ MulC(T c) :
        C(c)
    {
    }
    __host__ __device__ T operator()(T x)
    {
        return x * C;
    }
};

template<typename T>
struct line2col: public thrust::unary_function<T, T>
{
    T C;
    __host__ __device__ line2col(T C) :
            C(C)
    {
    }

    __host__ __device__ T operator()(T i)
    {
        return i / C;
    }
};

int main()
{
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    hipblasHandle_t hd;
    hiprandGenerator_t rng;
    hipblasCreate(&hd);
    hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT);

    const size_t m = 57, n = 5;
    const double c1 = 1.0;
    const double c0 = 0.0;

    thrust::device_vector<double> A(m * n);
    thrust::device_vector<double> B(m * n);
    thrust::device_vector<double> C(m * n);
    thrust::device_vector<double> sum1(1 * n);
    thrust::device_vector<double> sum2(1 * n);
    thrust::device_vector<double> one(m * n, 1);

    double* pA = thrust::raw_pointer_cast(&A[0]);
    double* pB = thrust::raw_pointer_cast(&B[0]);
    double* pSum1 = thrust::raw_pointer_cast(&sum1[0]);
    double* pSum2 = thrust::raw_pointer_cast(&sum2[0]);
    double* pOne = thrust::raw_pointer_cast(&one[0]);

    hiprandGenerateNormalDouble(rng, pA, A.size(), 0, 0.1);

    for(int i = 0; i < m; i++) {
        for(int j = 0; j < n; j++) {
            printf("%.8f", A[i*m + j]);
        }
        printf("\n");
    }

    const int count = 2;

    for (int i = 0; i < count; i++)
    {
        thrust::transform(A.begin(), A.end(), B.begin(), Exp());
        hipblasDgemv(hd, HIPBLAS_OP_T, m, n, &c1, pB, m, pOne, 1, &c0, pSum1, 1);
        thrust::transform(sum1.begin(), sum1.end(), sum1.begin(), Inv());
        hipblasDdgmm(hd, HIPBLAS_SIDE_RIGHT, m, n, pB, m, pSum2, 1, pB, m);
    }

    for (int i = 0; i < count; i++)
    {
        thrust::reduce_by_key(
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), line2col<int>(m)),
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), line2col<int>(m)) + A.size(),
                thrust::make_transform_iterator(A.begin(), Exp()),
                thrust::make_discard_iterator(),
                sum2.begin());
        thrust::transform(
                A.begin(), A.end(),
                thrust::make_permutation_iterator(
                        sum2.begin(),
                        thrust::make_transform_iterator(thrust::make_counting_iterator(0), line2col<int>(m))),
                C.begin(),
                thrust::divides<double>());
    }

    for (int i = 0; i < count; i++)
    {
        thrust::inclusive_scan_by_key(
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), line2col<int>(m)),
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), line2col<int>(m)) + A.size(),
                thrust::make_transform_iterator(A.begin(), Exp()),
                C.begin());
        thrust::copy(
                thrust::make_permutation_iterator(
                        C.begin() + m - 1,
                        thrust::make_transform_iterator(thrust::make_counting_iterator(0), MulC<int>(m))),
                thrust::make_permutation_iterator(
                        C.begin() + m - 1,
                        thrust::make_transform_iterator(thrust::make_counting_iterator(0), MulC<int>(m))) + n,
                sum2.begin());
        thrust::transform(
                A.begin(), A.end(),
                thrust::make_permutation_iterator(
                        sum2.begin(),
                        thrust::make_transform_iterator(thrust::make_counting_iterator(0), line2col<int>(m))),
                C.begin(),
                thrust::divides<double>());
    }

    hiprandDestroyGenerator(rng);
    hipblasDestroy(hd);

    return 0;
}